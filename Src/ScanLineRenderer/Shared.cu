#include "hip/hip_runtime.h"
#include <ScanLineRenderer/Shared.hpp>
#include <Base/CompileBegin.hpp>
#include <device_atomic_functions.h>
#include <Base/CompileEnd.hpp>

CUDAINLINE void cutTile(TileRef ref, unsigned int* cnt, TileRef* out, const unsigned int maxSize) {
    const auto by = ref.rect.z;
    constexpr auto step = 32U;
    const int xcnt = calcBlockSize(ref.rect.y - ref.rect.x, step);
    const int ycnt = calcBlockSize(ref.rect.w - ref.rect.z,step);
    const auto size= xcnt * ycnt;
    auto base = atomicAdd(cnt, size);
    if (base + size > maxSize)return;
    for (; ref.rect.x <= ref.rect.y; ref.rect.x += step) {
        for (ref.rect.z = by; ref.rect.z <= ref.rect.w; ref.rect.z += step) {
            out[base++] = ref;
        }
    }
}

GLOBAL void emitTile(const unsigned int size,unsigned int* cnt,READONLY(unsigned int) offset,
    READONLY(TileRef) in, TileRef* out, const unsigned int maxSize) {
    const auto id = getID();
    if(id>=size)return;
    const auto ref = in[id];
    if (ref.size== 5)cutTile(ref,cnt+5,out+offset[5],maxSize);
    else out[offset[ref.size] + atomicInc(cnt+ref.size,maxv)]=ref;
}

GLOBAL void sortTilesGPU(unsigned int* cnt, unsigned int* offset,unsigned int* tmp,
    TileRef* ref, TileRef* out, const unsigned int maxSize,const unsigned int maxOutSize){
    auto launchSize = cnt[6];
    if (cnt[6] > maxSize)launchSize = maxSize;

    offset[0] = 0;
    for (auto i = 1; i < 6; ++i)offset[i] = offset[i - 1] + cnt[i - 1];
    for (auto i = 0; i < 6; ++i)tmp[i] = 0;
    constexpr auto block = 1024U;
    run(emitTile, block, launchSize, tmp, offset, ref, out, maxOutSize);
    hipDeviceSynchronize();
    offset[5] += tmp[5];
}

std::pair<MemoryRef<unsigned int>, MemoryRef<TileRef>> sortTiles(CommandBuffer& buffer,
    const MemoryRef<unsigned int>& cnt, const MemoryRef<TileRef>& ref, const size_t refSize,
    const unsigned int maxSize) {
    auto sortedIdx = buffer.allocBuffer<TileRef>(refSize);
    auto tmp = buffer.allocBuffer<unsigned int>(6);
    auto offset = buffer.allocBuffer<unsigned int>(6);
    const unsigned int maxOutSize=sortedIdx.maxSize();
    buffer.callKernel(sortTilesGPU, cnt, offset, tmp, ref, sortedIdx, maxSize, maxOutSize);
    return { offset,sortedIdx };
}
