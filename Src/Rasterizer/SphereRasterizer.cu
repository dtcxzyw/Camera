#include "hip/hip_runtime.h"
#include <Rasterizer/SphereRasterizer.hpp>
#include <Core/DeviceFunctions.hpp>

DEVICEINLINE bool cmpMin(const float a, const float b) {
    return a < b;
}

DEVICEINLINE bool cmpMax(const float a, const float b) {
    return a > b;
}

template <typename Func, typename Cmp>
DEVICEINLINE float calcValue(const Func& func, const Cmp& cmp, float l, float r) {
    constexpr auto eps = 1e-3f;
    while (r - l >= eps) {
        const auto lm = (l * 2.0f + r) / 3.0f;
        const auto rm = (l + r * 2.0f) / 3.0f;
        if (cmp(func(lm), func(rm)))r = rm;
        else l = lm;
    }
    return func(l);
}

DEVICEINLINE bool calcSphereRange(const vec4& sphere, const float near, const float far, vec4& res) {
    const auto r2 = sphere.w * sphere.w;
    const auto begin = fmax(-far, sphere.z - sphere.w), end = fmin(-near, sphere.z + sphere.w);
    if (begin >= end)return false;
    res.x = calcValue([=](const float z) {
        const auto dz = z - sphere.z;
        return (sphere.x - sqrt(r2 - dz * dz)) / -z;
    }, cmpMin, begin, end);
    res.y = calcValue([=](const float z) {
        const auto dz = z - sphere.z;
        return (sphere.x + sqrt(r2 - dz * dz)) / -z;
    }, cmpMax, begin, end);
    res.z = calcValue([=](const float z) {
        const auto dz = z - sphere.z;
        return (sphere.y - sqrt(r2 - dz * dz)) / -z;
    }, cmpMin, begin, end);
    res.w = calcValue([=](const float z) {
        const auto dz = z - sphere.z;
        return (sphere.y + sqrt(r2 - dz * dz)) / -z;
    }, cmpMax, begin, end);
    return true;
}

GLOBAL void processSphereInfoKernel(const uint32_t size, READONLY(SphereDesc) in, SphereInfo* info,
    TileRef* ref, uint32_t* cnt, const vec4 scissor, const vec2 hsiz,
    const float near, const float far, const vec2 mul) {
    const auto id = getId();
    if (id >= size)return;
    const auto sphere = in[id];
    vec4 range;
    if (!calcSphereRange(*reinterpret_cast<const vec4*>(&sphere), near, far, range))return;
    const uvec4 rect = {
        fmax(scissor.x, (1.0f + range.x * mul.x) * hsiz.x - tileOffset),
        fmin(scissor.y, (1.0f + range.y * mul.x) * hsiz.x + tileOffset),
        fmax(scissor.z, (1.0f - range.w * mul.y) * hsiz.y - tileOffset),
        fmin(scissor.w, (1.0f - range.z * mul.y) * hsiz.y + tileOffset)
    };
    if (rect.x < rect.y & rect.z < rect.w) {
        const auto tsiz = calcTileSize(rect);
        deviceAtomicInc(cnt + tsiz, maxv);
        const auto wpos = deviceAtomicInc(cnt + 6, maxv);
        ref[wpos].id = wpos;
        ref[wpos].size = tsiz;
        ref[wpos].rect = rect;
        info[wpos].id = id;
        const auto pos = sphere.pos;
        info[wpos].info = {pos, 1.0f / sphere.radius};
        info[wpos].c = length2(Vector{pos}) - sphere.radius * sphere.radius;
    }
}

SphereProcessingResult processSphereInfo(CommandBuffer& buffer, const Span<SphereDesc>& spheres,
    const vec4 scissor, const vec2 hsiz, const float near, const float far,
    const vec2 mul) {
    auto cnt = buffer.allocBuffer<uint32_t>(7);
    buffer.memset(cnt);
    const auto info = buffer.allocBuffer<SphereInfo>(spheres.size());
    auto ref = buffer.allocBuffer<TileRef>(spheres.size());
    buffer.launchKernelLinear(makeKernelDesc(processSphereInfoKernel), spheres.size(), spheres, info, ref, cnt,
        scissor, hsiz, near, far, mul);
    const auto sortedSphere = sortTiles<Empty, unsigned char, emptyTileClipShader>
        (buffer, cnt, ref, spheres.size() * 2U + 2048U, spheres.size(), {}, {});
    cnt.reset();
    ref.reset();
    return SphereProcessingResult(sortedSphere.cnt, info, sortedSphere.array);
}
