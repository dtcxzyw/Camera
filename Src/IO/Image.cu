#include <IO/Image.hpp>
#include <Core/CompileBegin.hpp>
#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb_image_write.h>
#include <Core/CompileEnd.hpp>

struct ImageDeleter final {
    void operator()(float* ptr) const {
        stbi_image_free(ptr);
    }
};

using ImageHolder = std::unique_ptr<float, ImageDeleter>;

std::shared_ptr<BuiltinArray<RGBA>> loadRGBA(const std::string& path, Stream& stream) {
    stbi_set_flip_vertically_on_load(true);
    int w, h, channel;
    const ImageHolder image(stbi_loadf(path.c_str(), &w, &h, &channel, STBI_rgb_alpha));
    if (!image)throw std::runtime_error(stbi_failure_reason());
    auto res = std::make_shared<BuiltinArray<RGBA>>(uvec2(w, h));
    checkError(cudaMemcpyToArrayAsync(res->get(), 0, 0, image.get(), w * h * sizeof(RGBA)
        , hipMemcpyHostToDevice, stream.get()));
    stream.sync();
    return res;
}

std::shared_ptr<BuiltinMipmapedArray<RGBA>> loadMipmapedRGBA(const std::string& path, Stream& stream) {
    const auto src = loadRGBA(path, stream);
    return std::make_shared<BuiltinMipmapedArray<RGBA>>(*src, stream);
}

std::shared_ptr<BuiltinCubeMap<RGBA>> loadCubeMap(const std::function<std::string(size_t id)>& path, Stream& stream) {
    stbi_set_flip_vertically_on_load(false);
    std::shared_ptr<BuiltinCubeMap<RGBA>> res;
    std::vector<ImageHolder> freeList;
    for (size_t i = 0; i < 6; ++i) {
        auto rpath = path(i);
        int w, h, channel;
        ImageHolder image(stbi_loadf(rpath.c_str(), &w, &h, &channel, STBI_rgb_alpha));
        if (!image)throw std::runtime_error(stbi_failure_reason());
        if (!res)res = std::make_shared<BuiltinCubeMap<RGBA>>(w);
        hipMemcpy3DParms parm;
        memset(&parm, 0, sizeof(parm));
        parm.kind = hipMemcpyHostToDevice;
        parm.extent = make_hipExtent(w, w, 1);
        parm.srcPtr = make_hipPitchedPtr(image.get(), sizeof(RGBA) * w, w, h);
        parm.dstArray = res->get();
        parm.dstPos = make_hipPos(0, 0, i);
        checkError(hipMemcpy3DAsync(&parm, stream.get()));
        freeList.emplace_back(std::move(image));
    }
    stream.sync();
    return res;
}

std::pair<std::vector<float>, uvec2> loadDistribution2D(const std::string& path) {
    stbi_set_flip_vertically_on_load(true);
    int w, h, channel;
    const ImageHolder image(stbi_loadf(path.c_str(), &w, &h, &channel, STBI_grey));
    if (!image)throw std::runtime_error(stbi_failure_reason());
    return std::make_pair(std::vector<float>{image.get(), image.get() + w * h}, uvec2{w, h});
}

void saveHdr(const std::string& path, const float* pixel, const uvec2 size) {
    const auto res = stbi_write_hdr(path.c_str(), size.x, size.y, 3, pixel);
    if (res == 0)throw std::runtime_error("Failed to save.");
}
