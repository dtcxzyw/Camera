#include <GL/glew.h>
#include <Interaction/OpenGL.hpp>
#include <exception>

class GLContext final:Singletion {
private:
    bool mFlag;
    GLContext():mFlag(false) {
        if (!glfwInit())
            throw std::exception("Failed to initialize glfw.");
    }
    friend GLContext& getContext();
public:
    void makeContext(GLFWwindow* window) {
        GLFWwindow* current = nullptr;
        if (current != window) {
            glfwMakeContextCurrent(window);
            current = window;
            if (!mFlag) {
                glewExperimental = true;
                if (glewInit() != GLEW_NO_ERROR)
                    throw std::exception("Failed to initialize glew.");
                mFlag = true;
            }
        }
    }
    ~GLContext() {
        glfwTerminate();
    }
};

GLContext& getContext() {
    static GLContext context;
    return context;
}

void GLWindow::resize(uvec2 size) {
    if (size != mSize) {
        if (mRes) { 
            checkError(hipGraphicsUnregisterResource(mRes));
            mRes = 0;
        }
        glBindTexture(GL_TEXTURE_2D, mTexture);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F,size.x,size.y
            ,0,GL_RGBA,GL_UNSIGNED_BYTE,nullptr);
        checkError(hipGraphicsGLRegisterImage(&mRes, mTexture, GL_TEXTURE_2D
            , hipGraphicsRegisterFlagsSurfaceLoadStore));
        mSize = size;
    }
}

GLWindow::GLWindow():mRes(0) {
    auto& context=getContext();
    mWindow = glfwCreateWindow(800, 600, "OpenGL Viewer", nullptr, nullptr);
    if (!mWindow)
        throw std::exception("Failed to create a window.");
    context.makeContext(mWindow);
    glfwSwapInterval(0);
    glGenTextures(1, &mTexture);
    resize(size());
    glGenFramebuffers(1, &mFBO);
    glBindFramebuffer(GL_FRAMEBUFFER, mFBO);
    glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0,GL_TEXTURE_2D
        , mTexture, 0);
    auto res = glCheckFramebufferStatus(GL_FRAMEBUFFER);
    glBindFramebuffer(GL_FRAMEBUFFER,0);
    if(res!=GL_FRAMEBUFFER_COMPLETE)
        throw std::exception("Failed to create a FBO.");
}

void GLWindow::present(Pipeline& pipeline,const BuiltinRenderTarget<RGBA>& colorbuffer) {
    getContext().makeContext(mWindow);
    resize(colorbuffer.size());
    checkError(hipGraphicsMapResources(1, &mRes, pipeline.getId()));
    hipArray_t data;
    checkError(hipGraphicsSubResourceGetMappedArray(&data, mRes, 0, 0));
    checkError(cudaMemcpyArrayToArray(data, 0, 0, colorbuffer.get()
        , 0, 0, mSize.x*mSize.y * sizeof(RGBA)));
    checkError(hipGraphicsUnmapResources(1, &mRes, pipeline.getId()));
    auto frame = size();
    glBindFramebuffer(GL_READ_FRAMEBUFFER,mFBO);
    glBindFramebuffer(GL_DRAW_FRAMEBUFFER, 0);
    glBlitFramebuffer(0,0,frame.x,frame.y,0,0,mSize.x,mSize.y
        ,GL_COLOR_BUFFER_BIT,GL_NEAREST);
    glBindFramebuffer(GL_READ_FRAMEBUFFER, 0);
    glfwSwapBuffers(mWindow);
}

bool GLWindow::update() {
    glfwPollEvents();
    if (glfwWindowShouldClose(mWindow))
        return false;
    return true;
}

void GLWindow::resize(size_t width, size_t height) {
    glfwSetWindowSize(mWindow, width, height);
}

uvec2 GLWindow::size() const {
    int w, h;
    glfwGetFramebufferSize(mWindow, &w, &h);
    return { w,h };
}

GLWindow::~GLWindow() {
    glDeleteFramebuffers(1, &mFBO);
    if(mRes)checkError(hipGraphicsUnregisterResource(mRes));
    glDeleteTextures(1, &mTexture);
    glfwDestroyWindow(mWindow);
}
