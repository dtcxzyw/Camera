#include <Base/CompileBegin.hpp>
#include <GL/glew.h>
#include <Interaction/OpenGL.hpp>
#include <IMGUI/imgui_impl_glfw_gl3.h>
#include <Base/CompileEnd.hpp>

class GLContext final:Singletion {
private:
    bool mFlag;
    GLContext():mFlag(false) {
        if (!glfwInit())
            throw std::runtime_error("Failed to initialize glfw.");
        glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
        glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
        glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    }
    friend GLContext& getContext();
public:
    void makeContext(GLFWwindow* window) {
        static GLFWwindow* current = nullptr;
        if (current != window) {
            glfwMakeContextCurrent(window);
            glDisable(GL_FRAMEBUFFER_SRGB);
            current = window;
            if (!mFlag) {
                glewExperimental = true;
                if (glewInit() != GLEW_NO_ERROR)
                    throw std::runtime_error("Failed to initialize glew.");
                mFlag = true;
            }
        }
    }
    ~GLContext() {
        glfwTerminate();
    }
};

GLContext& getContext() {
    static GLContext context;
    return context;
}

GLWindow::GLWindow() {
    auto& context=getContext();
    mWindow = glfwCreateWindow(800, 600, "OpenGL Viewer", nullptr, nullptr);
    if (!mWindow)
        throw std::runtime_error("Failed to create a window.");
    context.makeContext(mWindow);
    glfwSwapInterval(0);
    glGenFramebuffers(1, &mFBO);
}

void GLWindow::present(Image& image) {
    getContext().makeContext(mWindow);
    glBindFramebuffer(GL_READ_FRAMEBUFFER, mFBO);
    glFramebufferTexture2D(GL_READ_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D
        , image.get(), 0);
    const auto isiz = image.size();
    const auto frameSize = size();
    glBindFramebuffer(GL_DRAW_FRAMEBUFFER, 0);
    glBlitFramebuffer(0, 0, frameSize.x, frameSize.y, 0, 0, isiz.x, isiz.y
        , GL_COLOR_BUFFER_BIT, GL_NEAREST);
    glBindFramebuffer(GL_READ_FRAMEBUFFER, 0);
}

void GLWindow::setVSync(const bool enable) {
    getContext().makeContext(mWindow);
    glfwSwapInterval(enable);
}

void GLWindow::swapBuffers() {
    glfwSwapBuffers(mWindow);
}

bool GLWindow::update() {
    getContext().makeContext(mWindow);
    glfwPollEvents();
    if (glfwWindowShouldClose(mWindow))
        return false;
    return true;
}

void GLWindow::resize(const uvec2 size) {
    glfwSetWindowSize(mWindow, size.x,size.y);
}

uvec2 GLWindow::size() const {
    int w, h;
    glfwGetFramebufferSize(mWindow, &w, &h);
    return { w,h };
}

GLWindow::~GLWindow() {
    glDeleteFramebuffers(1, &mFBO);
    glfwDestroyWindow(mWindow);
}

IMGUIWindow::IMGUIWindow() {
    if (!ImGui_ImplGlfwGL3_Init(mWindow,true))
        throw std::runtime_error("Failed to setup ImGui binding.");
}

void IMGUIWindow::newFrame() {
    getContext().makeContext(mWindow);
    ImGui_ImplGlfwGL3_NewFrame();
}

void IMGUIWindow::renderGUI() {
    getContext().makeContext(mWindow);
    const auto wsiz = size();
    glViewport(0, 0, wsiz.x, wsiz.y);
    ImGui::Render();
}

IMGUIWindow::~IMGUIWindow() {
    getContext().makeContext(mWindow);
    ImGui_ImplGlfwGL3_Shutdown();
}

Image::Image():mRes(nullptr) {
    glGenTextures(1, &mTexture);
}

Image::~Image() {
    if(mRes)checkError(hipGraphicsUnregisterResource(mRes));
    glDeleteTextures(1, &mTexture);
}

uvec2 Image::size() const {
    return mSize;
}

void Image::resize(const uvec2 size) {
    if (mSize != size) {
        if (mRes) {
            checkError(hipGraphicsUnregisterResource(mRes));
            mRes = nullptr;
        }
        glBindTexture(GL_TEXTURE_2D, mTexture);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, size.x, size.y
            , 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
        checkError(hipGraphicsGLRegisterImage(&mRes, mTexture, GL_TEXTURE_2D
            , hipGraphicsRegisterFlagsSurfaceLoadStore));
        mSize = size;
    }
}

hipArray_t Image::bind(const hipStream_t stream) {
    checkError(hipGraphicsMapResources(1, &mRes, stream));
    hipArray_t data;
    checkError(hipGraphicsSubResourceGetMappedArray(&data, mRes, 0, 0));
    return data;
}

void Image::unbind(const hipStream_t stream) {
    checkError(hipGraphicsUnmapResources(1, &mRes, stream));
}

GLuint Image::get() const {
    return mTexture;
}

