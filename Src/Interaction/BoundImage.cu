#include <Interaction/BoundImage.hpp>

void BoundImage::destoryRes() {
    if (mRes) {
        checkError(hipGraphicsUnregisterResource(mRes));
        mRes = nullptr;
    }
}

BoundImage::BoundImage(): mRes(nullptr) {}

uvec2 BoundImage::size() const {
    return mSize;
}

void BoundImage::resize(const uvec2 size) {
    if(mSize!=size) {
        destoryRes();
        mSize = size;
        reset();
    }
}

hipArray_t BoundImage::bind(const hipStream_t stream) {
    checkError(hipGraphicsMapResources(1, &mRes, stream));
    hipArray_t data;
    checkError(hipGraphicsSubResourceGetMappedArray(&data, mRes, 0, 0));
    return data;
}

void BoundImage::unbind(const hipStream_t stream) {
    checkError(hipGraphicsUnmapResources(1, &mRes, stream));
}

