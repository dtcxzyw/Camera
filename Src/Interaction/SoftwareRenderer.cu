#include <Interaction/SoftwareRenderer.hpp>
#include <Base/CompileBegin.hpp>
#include <IMGUI/imgui.h>
#include <Base/CompileEnd.hpp>
#include <ScanLineRenderer/TriangleRasterizer.hpp>
#include <Base/DataSet.hpp>
#include <ScanLineRenderer/IndexDescriptor.hpp>
#include <ScanLineRenderer/Buffer2D.hpp>

enum class VertOutAttr {
    TexCoord,
    Color
};

using VertOut = Args<VAR(VertOutAttr::TexCoord, vec2), VAR(VertOutAttr::Color, RGBA)>;

struct VertInfo final {
    ALIGN vec2 pos;
    ALIGN vec2 uv;
    ALIGN unsigned int col;
};

constexpr auto int2Float = 1.0f / 256.0f;

CUDAINLINE vec4 toRGBA(const unsigned int col) {
    return vec4{col & 0xff, (col >> 8) & 0xff, (col >> 16) & 0xff, col >> 24} * int2Float;
}

CUDAINLINE void vertShader(VertInfo in, const Empty&, vec3& cpos,
    VertOut& out) {
    cpos = {in.pos.x, in.pos.y, 1.0f};
    out.get<VertOutAttr::TexCoord>() = {in.uv.x, in.uv.y};
    out.get<VertOutAttr::Color>() = toRGBA(in.col);
}

struct FrameBufferInfo {
    BuiltinRenderTargetRef<RGBA8> color;
};

CUDAINLINE bool clipShader(unsigned int, vec3&, vec3&, vec3&, const BuiltinSamplerRef<float>&) {
    return true;
}

CUDAINLINE void colorShade(unsigned int id, ivec2 uv, float, const VertOut& in, const VertOut&, const VertOut&,
    const BuiltinSamplerRef<float>& texture, FrameBufferInfo& fbo) {
    const auto texAlpha = texture.get(in.get<VertOutAttr::TexCoord>());
    auto src = in.get<VertOutAttr::Color>();
    src.a *= texAlpha;
    const auto dst = vec4(fbo.color.get(uv)) * int2Float;
    const auto alpha = src.a, invAlpha = 1.0f - alpha;
    const auto col = vec3(src) * alpha + vec3(dst) * invAlpha;
    fbo.color.set(uv, RGBA8{clamp(col, 0.0f, 1.0f) * 255.0f, 255});
}

static bool isOverlap(vec2 a1, const vec2 a2, vec2 a3, vec2 b1, vec2 b2, vec2 b3) {
    const auto edgeFunction=[](const vec2 a,const vec2 b,const vec2 c) {
        return (c.x - a.x) * (b.y - a.y) - (c.y - a.y) * (b.x - a.x);
    };
    if (edgeFunction(a1, a2, a3) < 0.0f)std::swap(a1, a3);
    const auto test = [&](const vec2 a, const vec2 b, const vec2 c) {
        return edgeFunction(a, b, c) <= 0.0f;
    };
    const auto out=[&](const vec2 a,const vec2 b) {
        return test(a, b, b1) && test(a, b, b2) && test(a, b, b3);
    };
    return !(out(a1, a2) || out(a2, a3) || out(a3, a1));
}

void SoftwareRenderer::render(CommandBuffer& buffer, BuiltinRenderTarget<RGBA8>& renderTarget) {
    const auto drawData = ImGui::GetDrawData();
    #ifdef CAMERA_DEBUG
    if(!drawData->Valid)throw std::logic_error("This draw data is invalid.");
    #endif
    auto&& io = ImGui::GetIO();
    const int fbw = io.DisplaySize.x * io.DisplayFramebufferScale.x;
    const int fbh = io.DisplaySize.y * io.DisplayFramebufferScale.y;
    if (fbw == 0 || fbh == 0)return;
    drawData->ScaleClipRects(io.DisplayFramebufferScale);

    auto uni = buffer.allocConstant<BuiltinSamplerRef<float>>();
    buffer.memcpy(uni, [this](auto&& call) {
        const auto data = mSampler->toSampler();
        call(&data);
    });

    auto frameBuffer = buffer.allocConstant<FrameBufferInfo>();
    buffer.memcpy(frameBuffer, [rt = renderTarget.toTarget()](auto&& call) {
            FrameBufferInfo info;
            info.color = rt;
            call(&info);
        });

    auto vbo = buffer.allocBuffer<VertInfo>(drawData->TotalVtxCount);
    std::vector<VertInfo> vertData(vbo.size());
    {
        const auto mul = 2.0f / static_cast<vec2>(renderTarget.size());
        auto idx = 0U;
        for (auto i = 0; i < drawData->CmdListsCount; ++i) {
            for (const auto& vert : drawData->CmdLists[i]->VtxBuffer) {
                auto&& res = vertData[idx++];
                res.pos.x = vert.pos.x * mul.x, res.pos.y = -vert.pos.y * mul.y;
                res.pos.x -= 1.0f, res.pos.y += 1.0f;
                res.uv.x = vert.uv.x, res.uv.y = vert.uv.y;
                res.col = vert.col;
            }
        }
        buffer.memcpy(vbo, [buf = vertData](auto&& call) {
            call(buf.data());
        });
    }

    const auto vert = calcVertex<VertInfo, VertOut, Empty, vertShader>(buffer, vbo, nullptr);
    const auto vertBase = DataPtr<VertexInfo<VertOut>>{vert};
    auto vertBufferOffset = 0;

    auto ibo = buffer.allocBuffer<uvec3>(drawData->TotalIdxCount / 3);
    std::vector<uvec3> indexData(ibo.size());
    {
        auto idx = 0;
        for (auto i = 0; i < drawData->CmdListsCount; ++i) {
            auto&& idxBuf = drawData->CmdLists[i]->IdxBuffer;
            const auto idxSiz = idxBuf.size() / 3;
            for (auto j = 0; j < idxSiz; ++j)
                indexData[idx++] = { idxBuf[j * 3], idxBuf[j * 3 + 1], idxBuf[j * 3 + 2] };
        }
        buffer.memcpy(ibo, [buf = indexData](auto&& call) {
            call(buf.data());
        });
    }

    const auto testOverlap = [&vertData, &indexData](auto i, auto j, auto vertOffset, auto indexOffset) {
        const auto ii = indexData[i + indexOffset], ij = indexData[j + indexOffset];
        const auto base = vertData.data() + vertOffset;
        return isOverlap(base[ii[0]].pos, base[ii[1]].pos, base[ii[2]].pos,
            base[ij[0]].pos, base[ij[1]].pos, base[ij[2]].pos);
    };

    const auto iboBase = DataPtr<uvec3>{ibo};
    auto idxBufferOffset = 0;

    for (auto i = 0; i < drawData->CmdListsCount; ++i) {
        const auto cmdList = drawData->CmdLists[i];
        const auto vertPtr= vertBase + vertBufferOffset;

        for (auto j = 0; j < cmdList->CmdBuffer.size(); ++j) {
            const auto& cmd = cmdList->CmdBuffer[j];

            if (cmd.UserCallback)
                throw std::logic_error("Software renderer doesn't support user call back.");
            const vec4 scissor = {cmd.ClipRect.x, cmd.ClipRect.z, cmd.ClipRect.y, cmd.ClipRect.w};
            const auto faceCount = cmd.ElemCount / 3;
            const auto render = [&](auto base, auto size) {
                const auto idxPtr = iboBase + (idxBufferOffset + base);
                const auto index = makeIndexDescriptor<SeparateTrianglesWithIndex>(size, idxPtr.get());
                TriangleRenderingHistory history;
                history.reset(size, 65536U);
                renderTriangles<decltype(index), VertOut, BuiltinSamplerRef<float>,
                    FrameBufferInfo, clipShader, colorShade>(buffer,vertPtr, index, uni,frameBuffer,
                    renderTarget.size(), 0.5f, 1.5f, history, scissor, CullFace::None);
            };
            auto current = 0U;
            for (auto k = 0U; k < faceCount; ++k) {
                auto flag = false;
                for (auto l = current; l < k; ++l)
                    if (testOverlap(k, l, vertBufferOffset, idxBufferOffset)) {
                        flag = true;
                        break;
                    }

                if(flag){
                    render(current, k - current);
                    current = k;
                }
            }

            render(current, faceCount - current);

            idxBufferOffset += faceCount;
        }

        vertBufferOffset += cmdList->VtxBuffer.size();
    }
}

void SoftwareRenderer::init(Stream& resLoader) {
    //create font texture
    unsigned char* pixels;
    int width, height;
    ImGui::GetIO().Fonts->GetTexDataAsAlpha8(&pixels, &width, &height);
    mTexture = std::make_unique<BuiltinArray<float>>(uvec2{width, height});
    const auto size = width * height;
    PinnedBuffer<float> floatPixels(size);
    for (auto i = 0; i < size; ++i)
        floatPixels[i] = pixels[i] * int2Float;
    checkError(cudaMemcpyToArrayAsync(mTexture->get(), 0, 0, floatPixels.get(),
        size * sizeof(float), hipMemcpyHostToDevice, resLoader.get()));
    mSampler = std::make_unique<BuiltinSampler<float>>(mTexture->get());
    ImGui::GetIO().Fonts->SetTexID(mSampler.get());
    resLoader.sync();
}

void SoftwareRenderer::uninit() {
    mSampler.reset();
    mTexture.reset();
}
