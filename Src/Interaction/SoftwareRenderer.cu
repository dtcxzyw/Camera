#include <Interaction/SoftwareRenderer.hpp>
#include <Base/CompileBegin.hpp>
#include <IMGUI/imgui.h>
#include <Base/CompileEnd.hpp>
#include <ScanLineRenderer/TriangleRasterizer.hpp>
#include <Base/DataSet.hpp>
#include <ScanLineRenderer/IndexDescriptor.hpp>

enum class VertOutAttr {
    TexCoord,Color
};

using VertOut = Args<VAR(VertOutAttr::TexCoord, vec2), VAR(VertOutAttr::Color, RGBA)>;

constexpr auto int2Float = 1.0f / 255.0f;

vec4 toRGBA(const unsigned int col) {
    return vec4{ col & 0xff,(col >> 8) & 0xff,(col >> 16) & 0xff,col >> 24 }*int2Float;
}

struct VertInfo final{
    ALIGN vec2 pos;
    ALIGN vec2 uv;
    ALIGN vec4 col;
};

CUDAINLINE void vertShader(VertInfo in, const Empty&, vec3& cpos, 
    VertOut& out) {
    cpos = { in.pos.x,in.pos.y,1.0f };
    out.get<VertOutAttr::TexCoord>() = { in.uv.x,in.uv.y };
    out.get<VertOutAttr::Color>() = in.col;
}

CUDAINLINE bool clipShader(unsigned int, vec3&, vec3&, vec3&, const BuiltinSamplerGPU<float>&) {
    return true;
}

CUDAINLINE void fragShader(unsigned int, ivec2 uv, float, const VertOut& in, const VertOut&, const VertOut&,
    const BuiltinSamplerGPU<float>& texture, BuiltinRenderTargetGPU<RGBA8>& fbo) {
    const auto texAlpha = texture.get(in.get<VertOutAttr::TexCoord>());
    auto src = in.get<VertOutAttr::Color>();
    src.a *= texAlpha;
    const auto dst = vec4(fbo.get(uv))*int2Float;
    const auto alpha = src.a, invAlpha = 1.0f - alpha;
    const auto col = src*alpha + dst * invAlpha;
    const auto res = clamp(col, 0.0f, 1.0f);
    //const auto res = clamp(vec4{ vec3{src},1.0f }, 0.0f, 1.0f);
    fbo.set(uv, RGBA8{res*253.0f});
}

void SoftwareRenderer::renderDrawLists(ImDrawData* drawData,CommandBuffer& buffer,
    BuiltinRenderTarget<RGBA8>& renderTarget) {
    auto&& io = ImGui::GetIO();
    const int fbw = io.DisplaySize.x * io.DisplayFramebufferScale.x;
    const int fbh = io.DisplaySize.y * io.DisplayFramebufferScale.y;
    if (fbw == 0 || fbh == 0)return;
    drawData->ScaleClipRects(io.DisplayFramebufferScale);

    auto uni=buffer.allocConstant<BuiltinSamplerGPU<float>>();
    buffer.memcpy(uni, [this](auto&& call) {
        const auto data = mSampler->toSampler();
        call(&data);
    });

    auto frameBuffer = buffer.allocConstant<BuiltinRenderTargetGPU<RGBA8>>();
    buffer.memcpy(frameBuffer,[rt=renderTarget.toTarget()](auto&& call) {
        call(&rt);
    });

    auto vbo = buffer.allocBuffer<VertInfo>(drawData->TotalVtxCount);
    {
        const auto mul = 2.0f / static_cast<vec2>(renderTarget.size());
        std::vector<VertInfo> tmp(vbo.size());
        auto idx = 0U;
        for(auto i = 0; i < drawData->CmdListsCount; ++i){
            const auto cmdList = drawData->CmdLists[i];
            for (const auto& vert : cmdList->VtxBuffer) {
                auto&& res = tmp[idx++];
                res.pos.x = vert.pos.x*mul.x, res.pos.y = -vert.pos.y*mul.y;
                res.pos.x -= 1.0f, res.pos.y += 1.0f;
                res.uv.x = vert.uv.x, res.uv.y = vert.uv.y;
                res.col = toRGBA(vert.col);
            }
        }
        buffer.memcpy(vbo, [buf = std::move(tmp)](auto&& call) {
            call(buf.data());
        });
    }

    const auto vert = calcVertex<VertInfo, VertOut, Empty, vertShader>(buffer, vbo, nullptr);
    const auto vboBase = DataPtr<VertexInfo<VertOut>>{ vert };
    auto vertBufferOffset = 0;

    auto ibo = buffer.allocBuffer<uvec3>(drawData->TotalIdxCount / 3);
    {
        std::vector<uvec3> tmp(ibo.size());
        for (auto i = 0; i < drawData->CmdListsCount; ++i){
            auto&& idxBuf = drawData->CmdLists[i]->IdxBuffer;
            for (auto j = 0; j < tmp.size(); ++j)
                tmp[j] = { idxBuf[j * 3],idxBuf[j * 3 + 1],idxBuf[j * 3 + 2] };
            buffer.memcpy(ibo, [buf = std::move(tmp)](auto&& call) {
                call(buf.data());
            });
        }
    }

    const auto iboBase = DataPtr<uvec3>{ ibo };
    auto idxBufferOffset = 0;

    for (auto i = 0; i < drawData->CmdListsCount; ++i) {
        const auto cmdList = drawData->CmdLists[i];

        for (auto j = 0; j < cmdList->CmdBuffer.size(); ++j) {
            const auto& pcmd = cmdList->CmdBuffer[j];
            if (pcmd.UserCallback)
                throw std::logic_error("Software renderer doesn't support user call back.");
            const vec4 scissor = { pcmd.ClipRect.x,pcmd.ClipRect.z, pcmd.ClipRect.y, pcmd.ClipRect.w };
            const auto faceCount = pcmd.ElemCount / 3;
            const auto index = makeIndexDescriptor<SeparateTrianglesWithIndex>(faceCount,
                ibo, faceCount);
            TriangleRenderingHistory history;
            history.reset(faceCount, 65536U);
            renderTriangles<decltype(index), VertOut, BuiltinSamplerGPU<float>,
                BuiltinRenderTargetGPU<RGBA8>, clipShader, fragShader>(buffer, 
                    vboBase + vertBufferOffset, index, uni,frameBuffer, renderTarget.size(), 0.5f, 1.5f, 
                    history, scissor, CullFace::None);
            idxBufferOffset += faceCount;
        }

        vertBufferOffset += cmdList->VtxBuffer.size();
    }
}

void SoftwareRenderer::init(Stream& resLoader) {
    //create font texture
    unsigned char* pixels;
    int width, height;
    ImGui::GetIO().Fonts->GetTexDataAsAlpha8(&pixels, &width, &height); 
    mTexture = std::make_unique<BuiltinArray<float>>(uvec2{width,height});
    const auto size = width * height;
    PinnedBuffer<float> floatPixels(size);
    for (auto i = 0; i < size; ++i)
        floatPixels[i] = pixels[i] * int2Float;
    checkError(cudaMemcpyToArrayAsync(mTexture->get(),0,0,floatPixels.get(),
        size*sizeof(float),hipMemcpyHostToDevice, resLoader.get()));
    mSampler = std::make_unique<BuiltinSampler<float>>(mTexture->get());
    resLoader.sync();
}

void SoftwareRenderer::uninit() {
    mSampler.reset();
    mTexture.reset();
}

void SoftwareRenderer::render(CommandBuffer& buffer,BuiltinRenderTarget<RGBA8>& renderTarget) {
    ImGui::Render();
    renderDrawLists(ImGui::GetDrawData(), buffer, renderTarget);
}
