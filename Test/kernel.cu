#include "hip/hip_runtime.h"
#include <ScanLineRenderer/ScanLineRenderer.hpp>
#include "kernel.hpp"
#include <hip/device_functions.h>
#include <device_atomic_functions.h>
#include <PBR/Dist.hpp>

CUDAInline vec3 toPos(vec3 p,vec2 mul) {
    return { p.x*mul.x,p.y*mul.y,-p.z };
}

CUDAInline bool CS(unsigned int, vec3& pa, vec3& pb, vec3& pc, Uniform u) {
    pa = toPos(pa, u.mul);
    pb = toPos(pb, u.mul);
    pc = toPos(pc, u.mul);
    return true;
}

CUDAInline void VS(VI in, Uniform uniform, vec3& cpos, OI& out) {
    auto wp =uniform.Msky*vec4(in.pos, 1.0f);
    out.get<pos>() = in.pos;
    cpos = mat4(mat3(uniform.V))*wp;
}

CUDAInline void setSkyPoint(unsigned int,ivec2 uv,float, OI, Uniform, FrameBufferGPU& fbo) {
    fbo.depth.set(uv, 0xfffffffc);
}

CUDAInline void drawSky(unsigned int triID,ivec2 uv, float,OI out, Uniform uniform, FrameBufferGPU& fbo) {
    if (fbo.depth.get(uv) == 0xfffffffc) {
        auto p =out.get<pos>();
        fbo.color.set(uv, uniform.sampler.getCubeMap(p));
        //fbo.color.set(uv, uniform.sampler.get(calcHDRUV(p)));
    }
}

CUDAInline void VSM(VI in, Uniform uniform, vec3& cpos, OI& out) {
    auto wp = uniform.M*vec4(in.pos, 1.0f);
    out.get<pos>() = wp;
    out.get<normal>() = uniform.invM*in.normal;
    out.get<tangent>() = uniform.invM*in.tangent;
    cpos = uniform.V*wp;
}

CUDAInline bool CSM(unsigned int id, vec3& pa, vec3& pb, vec3& pc, Uniform u) {
    pa = toPos(pa, u.mul);
    pb = toPos(pb, u.mul);
    pc = toPos(pc, u.mul);
    return u.cache.query(id);
}

constexpr float maxdu = std::numeric_limits<unsigned int>::max();

CUDAInline void setPoint(unsigned int, ivec2 uv, float z, OI, Uniform, FrameBufferGPU& fbo) {
    fbo.depth.set(uv, z*maxdu);
}

CUDAInline void drawPoint(unsigned int triID, ivec2 uv, float z, OI out, Uniform uniform, FrameBufferGPU& fbo) {
    if (fbo.depth.get(uv) == static_cast<unsigned int>(z*maxdu)) {
        uniform.cache.record(triID);
        auto p = out.get<pos>();
        vec3 N =normalize(out.get<normal>());
        vec3 X = normalize(out.get<tangent>());
        vec3 Y = normalize(cross(N, X));
        auto off = uniform.lp - p;
        auto dis2 = length2(off);
        auto dis = sqrt(dis2);
        auto L = off/dis;
        auto V = normalize(uniform.cp - p);
        auto F = disneyBRDF(L, V, N, X, Y, uniform.arg);
        auto ref = reflect(-V,N);
        auto lc = uniform.lc +100.0f*vec3(uniform.sampler.getCubeMap(ref));
        auto res = lc*F*(distUE4(dis2,uniform.r*uniform.r)*dot(N, L));
        fbo.color.set(uv, { res,1.0f });
    }
}

CUDAInline void post(ivec2 NDC, PostUniform uni, BuiltinRenderTargetGPU<RGBA> out) {
    RGB c = uni.in.color.get(NDC);
    auto lum = luminosity(c);
    if (uni.in.depth.get(NDC)<0xfffffffc) {
        if (lum > 0.0f) {
            atomicAdd(&uni.sum->first, log(lum));
            atomicInc(&uni.sum->second, maxv);
        }
        c = ACES(c, *uni.lum);
    }
    c = pow(c, vec3(1.0f / 2.2f));
    NDC.y = uni.in.mSize.y - 1 - NDC.y;
    out.set(NDC, { c,1.0f });
}

CALLABLE void updateLum(PostUniform uniform) {
    *uniform.lum=calcLum(uniform.sum->first/(uniform.sum->second+1));
}

template<VSF<VI,OI,Uniform> vs,TCSF<Uniform> cs, FSF<OI, Uniform, FrameBufferGPU> ds,
    FSF<OI,Uniform,FrameBufferGPU> fs>
void renderMesh(const StaticMesh& model , const MemoryRef<Uniform>& uniform,
    FrameBufferCPU & fbo, Camera::RasterPosConverter converter,CullFace mode,
   TriangleRenderingHistory& history, CommandBuffer & buffer) {
    auto vert = calcVertex<VI, OI, Uniform, vs>(buffer, model.mVert, uniform);
    renderTriangles<SharedIndex, OI, Uniform, FrameBufferGPU,cs, ds, fs>(buffer, vert,
        model.mIndex, uniform, fbo.getData(buffer), fbo.size,
        converter.near, converter.far,history,mode);
}

void kernel(const StaticMesh& model,TriangleRenderingHistory& mh,
    const StaticMesh& skybox,TriangleRenderingHistory& sh,
    const MemoryRef<Uniform>& uniform, FrameBufferCPU & fbo, float* lum,
    Camera::RasterPosConverter converter, CommandBuffer & buffer) {
    fbo.colorRT->clear(buffer, vec4{ 0.0f,0.0f,0.0f,1.0f });
    fbo.depthBuffer->clear(buffer);
    renderMesh<VS,CS, setSkyPoint, drawSky>(skybox, uniform, fbo, converter,CullFace::Front,sh,buffer);
    renderMesh<VSM,CSM,setPoint,drawPoint>(model,uniform,fbo,converter,CullFace::Back,mh,buffer);
    auto puni = buffer.allocConstant<PostUniform>();
    auto sum = buffer.allocBuffer<std::pair<float, unsigned int>>();
    buffer.memset(sum);
    auto punidata = buffer.makeLazyConstructor<PostUniform>(fbo.data,lum,sum);
    buffer.memcpy(puni, [punidata,&buffer](auto call) {
        auto pd = punidata;
        auto data=pd.get(buffer);
        call(&data);
    });
    ResourceRef<BuiltinRenderTargetGPU<RGBA>> image
        = std::make_shared<ImageResource>(buffer,fbo.image);
    renderFullScreen<PostUniform,BuiltinRenderTargetGPU<RGBA>, post>(buffer, puni, image, 
        fbo.size);
    buffer.callKernel(updateLum,punidata);
}
