#include "hip/hip_runtime.h"
#include <ScanLineRenderer/ScanLineRenderer.hpp>
#include "kernel.hpp"
#include <PBR/BRDF.hpp>
#include <hip/device_functions.h>
#include <ScanLineRenderer/Primitive.hpp>

CUDA void GS(VI* in, Uniform uniform,Queue<VI,3> out) {
    auto dab = in[0].pos - in[1].pos, dcb = in[2].pos - in[1].pos;
    auto off = normalize(cross(dcb,dab))*uniform.off;
    for (int i = 0; i < 3; ++i)in[i].pos += off;
    out.push(in);
}

CUDA void VS(VI in, Uniform uniform, vec4& NDC, OI& out) {
    auto wp =uniform.M*vec4(in.pos, 1.0f);
    out.get<pos>() = wp;
    out.get<normal>() = uniform.invM*in.normal;
    //out.get<bin>() = uniform.invM*in.tangent;
    NDC = uniform.VP*wp;
}

constexpr float maxdu = std::numeric_limits<unsigned int>::max();

CUDA void setPoint(ivec2 uv,float z, OI out, Uniform uniform, FrameBufferGPU& fbo) {
    fbo.depth.set(uv, z*maxdu);
}

CUDA void drawPoint(ivec2 uv, float z,OI out, Uniform uniform, FrameBufferGPU& fbo) {
    if (fbo.depth.get(uv) ==static_cast<unsigned int>(z*maxdu)) {
        auto p = out.get<pos>();
        vec3 nd =normalize(out.get<normal>());
        //auto tangent = normalize(out.get<bin>());
        auto in = uniform.dir;
        auto out = normalize(uniform.cp - p);
        auto h = calcHalf(in, out);
        auto ndi = dot(nd, in);
        auto ndo = dot(nd, out);
        auto idh = dot(in, h);
        auto diff =uniform.color * disneyDiffuse(ndi,ndo,idh,uniform.roughness);
        auto D = GGXD(ndo, uniform.roughness);
        auto F = fresnelSchlick(uniform.f0,ndo);
        auto G = smithG(ndi, ndo, uniform.roughness);
        auto w =diff+ cookTorrance(D, F, G, ndi, ndo);
        auto res = uniform.color*uniform.lc*w*fmax(ndi,0.0f);
        //auto res = uniform.sampler.get(in,out,h,nd,tangent)*uniform.lc;
        fbo.color.set(uv, {res,1.0f });
    }
}

CUDA void post(ivec2 NDC, PostUniform uni, BuiltinRenderTargetGPU<RGBA> out) {
    RGB c = uni.in.color.get(NDC);
    auto lum = luminosity(c);
    if(lum>0.0f)atomicAdd(uni.sum,log(lum));
    c = ACES(c,uni.lum);
    NDC.y = uni.in.mSize.y- 1 - NDC.y;
    out.set(NDC, { c,1.0f });
}

void kernel(DataViewer<VI> vbo, DataViewer<uvec3> ibo, const Uniform* uniform
    , FrameBufferCPU& fbo, const PostUniform* puni,
    BuiltinRenderTargetGPU<RGBA> dest, Pipeline& pipeline) {
    fbo.colorRT->clear(pipeline,vec4{ 0.0f,0.0f,0.0f,1.0f });
    fbo.depthBuffer->clear(pipeline);
    auto prim = genPrimitive<3,3,SharedIndex, VI, Uniform, GS>(pipeline, vbo,ibo,uniform);
    auto vert = calcVertex<VI, OI, Uniform, VS>(pipeline, prim,uniform,fbo.size);
    renderTriangles<UniqueIndex, OI, Uniform, FrameBufferGPU, setPoint, drawPoint>
        (pipeline, vert, prim.size()/3, uniform, fbo.dataGPU.get(), fbo.size);
    renderFullScreen<PostUniform, decltype(dest), post>(pipeline,puni,dest,fbo.size);
}

