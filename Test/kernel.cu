#include "hip/hip_runtime.h"
#include <PostProcess/ToneMapping.hpp>
#include <Rasterizer/PostProcess.hpp>
#include "kernel.hpp"
#include <Rasterizer/SphereRasterizer.hpp>
#include <Rasterizer/IndexDescriptor.hpp>
#include <Texture/Noise.hpp>

CUDAINLINE vec3 toPos(const vec3 p, const Uniform& u) {
    return {p.x * u.mul.x, p.y * u.mul.y, -p.z};
}

CUDAINLINE void setDepth(unsigned int& data, const unsigned int val) {
    atomicMin(&data, val);
}

CUDAINLINE bool CS(unsigned int, vec3& pa, vec3& pb, vec3& pc, const Uniform& u) {
    pa = toPos(pa, u);
    pb = toPos(pb, u);
    pc = toPos(pc, u);
    return true;
}

CUDAINLINE void VS(VI in, const Uniform& uniform, vec3& cpos, OI& out) {
    const auto wp = uniform.Msky * vec4(in.pos, 1.0f);
    out.get<Pos>() = in.pos;
    cpos = mat4(mat3(uniform.V)) * wp;
}

CUDAINLINE void drawSky(unsigned int, ivec2 uv, float, const OI& out, const OI&, const OI&,
    const Uniform& uniform, FrameBufferRef& fbo) {
    if (fbo.depth.get(uv) == 0xffffffff) {
        const vec3 p = out.get<Pos>();
        fbo.color.set(uv, uniform.sampler.getCubeMap(p));
    }
}

CUDAINLINE void VSM(VI in, const Uniform& uniform, vec3& cpos, OI& out) {
    const auto wp = uniform.M * vec4(in.pos, 1.0f);
    out.get<Pos>() = wp;
    out.get<Normal>() = uniform.normalMat * in.normal;
    out.get<Tangent>() = uniform.normalMat * in.tangent;
    cpos = uniform.V * wp;
}

CUDAINLINE bool CSM(unsigned int, vec3& pa, vec3& pb, vec3& pc, const Uniform& u) {
    pa = toPos(pa, u);
    pb = toPos(pb, u);
    pc = toPos(pc, u);
    return true;
}

constexpr float maxdu = std::numeric_limits<unsigned int>::max();

CUDAINLINE void setModel(unsigned int, ivec2 uv, float z, const OI&, const OI&, const OI&,
    const Uniform&, FrameBufferRef& fbo) {
    setDepth(fbo.depth.get(uv), z * maxdu);
}

CUDAINLINE vec3 shade(const vec3 p, const vec3 N, const vec3 X, const vec3 Y, 
    const Uniform& uniform) {
    const auto sample = uniform.light.sample({}, p);
    const auto V = normalize(uniform.cp - p);
    const auto F = disneyBRDF(sample.wi, V, N, X, Y, uniform.arg);
    const auto ref = reflect(-V, N);
    const auto lc = uniform.light.sample({}, p).illumination +
        RGBSpectrum(uniform.sampler.getCubeMap(ref));
    return lc * F * fabs(dot(N, sample.wi));
}

CUDAINLINE void drawModel(unsigned int, ivec2 uv, float z, const OI& out, const OI&,
    const OI&, const Uniform& uniform, FrameBufferRef& fbo) {
    if (fbo.depth.get(uv) == static_cast<unsigned int>(z * maxdu)) {
        const vec3 p = out.get<Pos>();
        const vec3 N = normalize(out.get<Normal>());
        const vec3 T = normalize(out.get<Tangent>());
        const auto X = normalize(T - dot(T, N) * N);
        const auto Y = normalize(cross(X, N));
        fbo.color.set(uv, { shade(p,N,X,Y,uniform), 1.0f });
    }
}

CUDAINLINE void setPoint(unsigned int, ivec2 uv, float z, const OI&,
    const Uniform&, FrameBufferRef& fbo) {
    setDepth(fbo.depth.get(uv), z * maxdu);
}

CUDAINLINE void drawPoint(unsigned int, ivec2 uv, float z, const OI&,
    const Uniform&, FrameBufferRef& fbo) {
    if (fbo.depth.get(uv) == static_cast<unsigned int>(z * maxdu))
        fbo.color.set(uv, {1.0f, 1.0f, 1.0f, 1.0f});
}

CUDAINLINE void post(ivec2 NDC, const PostUniform& uni, BuiltinRenderTargetRef<RGBA8> out) {
    RGBSpectrum c = uni.in.color.get(NDC);
    if (uni.in.depth.get(NDC) < 0xffffffff) {
        const auto lum = c.lum();
        if (lum > 0.0f) {
            atomicAdd(&uni.sum->first, log(lum));
            atomicInc(&uni.sum->second, maxv);
        }
        c = ACES(c, *uni.lum);
    }
    c = clamp(pow(c, vec3(1.0f / 2.2f)), 0.0f, 1.0f);
    const RGBA8 color = {c * 255.0f, 255};
    out.set(NDC, color);
}

GLOBAL void updateLum(const PostUniform uniform) {
    *uniform.lum = calcLum(uniform.sum->first / (uniform.sum->second + 1));
}

template <VertShader<VI, OI, Uniform> VertFunc, TriangleClipShader<Uniform> ClipFunc, 
    FragmentShader<OI, Uniform, FrameBufferRef>... FragFunc>
void renderMesh(const StaticMesh& model, const Span<Uniform>& uniform,
    const Span<FrameBufferRef>& frameBuffer, const uvec2 size,
    const PinholeCamera::RasterPosConverter converter,
    const CullFace mode, RenderingContext& context, const vec4 scissor,
    CommandBuffer& buffer) {
    auto vert = calcVertex<VI, OI, Uniform, VertFunc>(buffer, buffer.useAllocated(model.vert),
        uniform, context.get());
    const auto index = makeIndexDescriptor<SeparateTrianglesWithIndex>(model.index.size(),
        buffer.useAllocated(model.index));
    renderTriangles<decltype(index), OI, Uniform, FrameBufferRef, ClipFunc,
        emptyTriangleTileClipShader<Uniform>, VersionComparer, FragFunc...>(buffer, vert, index,
            uniform, frameBuffer, size, converter.near, converter.far, scissor, context.triContext, 
            context.vertCounter.get(), mode);
}

CUDAINLINE vec4 vsSphere(vec4 sp, const Uniform& uniform) {
    return calcCameraSphere(sp, uniform.V);
}

CUDAINLINE void setSpherePoint(unsigned int, ivec2 uv, float z, vec3, vec3, float, bool,
    vec3, vec3, const Uniform&, FrameBufferRef& fbo) {
    setDepth(fbo.depth.get(uv), z * maxdu);
}

CUDAINLINE void drawSpherePoint(unsigned int, ivec2 uv, float z, vec3 p, vec3 dir, float invr,
    bool inSphere, vec3 dpdx, vec3 dpdy, const Uniform& u, FrameBufferRef& fbo) {
    if (fbo.depth.get(uv) == static_cast<unsigned int>(z * maxdu)) {
        const vec3 pos = u.invV * vec4(p, 1.0f);
        const auto modelDir = u.normalInvV*dir;
        const auto normalizedDir = modelDir * invr;
        const auto N = calcSphereNormal(normalizedDir, inSphere);
        const auto Y = calcSphereBiTangent(N);
        const auto X = calcSphereTangent(N, Y);
        auto res = shade(pos, N, X, Y, u);
        const auto octaves = calcOctavesAntiAliased(u.normalInvV*dpdx, u.normalInvV*dpdy);
        res *= marble(modelDir, 1.0f, 0.5f, octaves) + 0.5f;
        fbo.color.set(uv, { res, 1.0f });
    }
}

void kernel(const StaticMesh& model, RenderingContext& mc,
    const StaticMesh& skybox, RenderingContext& sc,
    const MemorySpan<vec4>& spheres,
    const Span<Uniform>& uniform, FrameBuffer& fbo, float* lum,
    const PinholeCamera::RasterPosConverter converter, CommandBuffer& buffer) {
    fbo.colorRT->clear(buffer, vec4{ 0.0f, 0.0f, 0.0f, 1.0f });
    Buffer2D<unsigned int> depth(buffer, fbo.size);
    depth.clear(0xff);
    const auto frameBuffer = fbo.getData(buffer, depth);
    const vec4 scissor = {0.0f, fbo.size.x, 0.0f, fbo.size.y};
    renderMesh<VSM, CSM, setModel, drawModel>(model, uniform, frameBuffer, fbo.size,
        converter, CullFace::Back, mc, scissor, buffer);
    renderSpheres<Uniform, FrameBufferRef, vsSphere, setSpherePoint, drawSpherePoint>(buffer,
        buffer.useAllocated(spheres), uniform, frameBuffer, fbo.size, converter.near, converter.far, 
        converter.mul, scissor);
    renderMesh<VS, CS, drawSky>(skybox, uniform, frameBuffer, fbo.size, converter,
        CullFace::Front, sc, scissor, buffer);
    const auto puni = buffer.allocConstant<PostUniform>();
    const auto sum = buffer.allocBuffer<std::pair<float, unsigned int>>();
    buffer.memset(sum);
    const auto depthBufferRef = depth.toBuffer();
    const auto fboData = buffer.makeLazyConstructor<FrameBufferRef>(fbo.data, depthBufferRef);
    auto punidata = buffer.makeLazyConstructor<PostUniform>(fboData, lum, sum);
    auto&& manager = buffer.getResourceManager();
    buffer.memcpy(puni, [punidata,&manager](auto call) {
        auto pd = punidata;
        auto data = pd.get(manager);
        call(&data);
    });
    renderFullScreen<PostUniform, BuiltinRenderTargetRef<RGBA8>, post>(buffer, puni,
        fbo.postRT->toTarget(), fbo.size);
    buffer.callKernel(updateLum, punidata);
}
