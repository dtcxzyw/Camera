#include "hip/hip_runtime.h"
#include <cstdio>
#include <system_error>
#include "kernel.hpp"
#include <Interaction/OpenGL.hpp>
#include <thread>
using namespace std::chrono_literals;

int main() {
    getEnvironment().init();
    try {
        StaticMesh model;
        model.load("Res/bunny.obj");
        printf("vertices %d ,triangles: %d\n", static_cast<int>(model.mVert.size()),
            static_cast<int>(model.mIndex.size()));

        MERLBRDFData brdf("Res/steel.binary");

        FrameBufferCPU FB;
        GLWindow window;
        Pipeline pipeline;
        glm::mat4 V = lookAt({ 10.0f,0.0f,0.0f }, vec3{ 0.0f,0.0f,0.0f }, { 0.0f,1.0f,0.0f });
        glm::mat4 M;
        M = scale(M, vec3(1.0f, 1.0f, 1.0f)*10.0f);
        float t = glfwGetTime(),lum=1.0f,last=1.0f;
        Constant<Uniform> uniform;
        Constant<PostUniform> puni;
        while (window.update()) {
            auto size = window.size();
            if (size.x == 0 || size.y == 0) {
                std::this_thread::sleep_for(1ms);
                continue;
            }
            FB.resize(size.x, size.y,pipeline);
            float w = size.x, h = size.y;
            glm::mat4 P = perspectiveFov(radians(45.0f), w, h, 1.0f, 20.0f);
            float now = glfwGetTime();
            float delta = now - t;
            M = rotate(M, delta*0.2f, { 0.0f,1.0f,0.0f });
            printf("\r%.2f ms          ", delta*1000.0f);
            t = now;
            Uniform u;
            u.VP = P*V;
            u.M = M;
            u.invM = mat3(transpose(inverse(M)));
            u.lc = vec3(5.0f);
            u.color = {1.0f,0.84f,0.0f};
            u.cp = { 10.0f,4.0f,0.0f };
            u.dir = normalize(u.cp);
            u.roughness = 0.5f;
            u.f0 = { 1.00f, 0.71f, 0.29f };
            u.sampler = brdf.toSampler();
            uniform.set(u, pipeline);
            BuiltinRenderTarget<RGBA> RT(window.map(pipeline,size),size);
            auto sum = allocBuffer<float>();
            *sum = 0.0f;
            PostUniform post;
            post.in = FB.data;
            auto tw = powf(0.2f, delta);
            auto nlum = lum*(1.0f - tw) + last*tw;
            last = nlum;
            post.lum =calcLum(nlum);
            post.sum = sum.begin();
            puni.set(post, pipeline);
            kernel(model.mVert, model.mIndex, uniform.get(), FB, puni.get(),RT.toTarget(), pipeline);
            window.unmapAndPresent(pipeline);
            pipeline.sync();
            lum =*sum/(w*h);
        }
    }
    catch (const std::exception& e) {
        puts("Catched an error:");
        puts(e.what());
        system("pause");
    }
    return 0;
}
