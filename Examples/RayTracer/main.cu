#include "hip/hip_runtime.h"
#include <IO/Model.hpp>
#include <Core/Environment.hpp>
#include <RayTracer/Scene.hpp>
#include <Core/Constant.hpp>
#include <Camera/PinholeCamera.hpp>
#include <RayTracer/BVH.hpp>
#include <Light/LightWrapper.hpp>
#include <Light/DeltaPositionLight.hpp>
#include <Spectrum/SpectrumConfig.hpp>
#include <RayTracer/RenderingAPI.hpp>
#include <Core/CompileBegin.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <Core/CompileEnd.hpp>
#include <RayTracer/Film.hpp>
#include <Camera/RayGeneratorWrapper.hpp>
#include <IO/Image.hpp>
#include <Material/MaterialWrapper.hpp>
#include <RayTracer/Integrators/Path.hpp>
#include <RayTracer/Integrators/Whitted.hpp>

using namespace std::chrono_literals;

struct App final : Uncopyable {
private:
    PinholeCamera mCamera;
    std::vector<std::unique_ptr<BvhForTriangle>> mBvh;
    std::vector<std::unique_ptr<Constant<BvhForTriangleRef>>> mBvhRef;
    std::vector<MemorySpan<LightWrapper>> mLight;
    std::unique_ptr<SceneDesc> mScene;
    std::unique_ptr<PathIntegrator> mIntegrator;
    MemorySpan<MaterialWrapper> mMaterial;
public:
    void addModel(Stream& resLoader, std::vector<Primitive>& primitives,
        const glm::mat4& trans, const std::string& path, MaterialWrapper* material) {
        StaticMesh mesh(path);
        mBvh.emplace_back(std::make_unique<BvhForTriangle>(mesh, 32U, resLoader));
        mBvhRef.emplace_back(std::make_unique<Constant<BvhForTriangleRef>>());
        mBvhRef.back()->set(mBvh.back()->getRef(), resLoader);
        primitives.emplace_back(Primitive{Transform(trans), mBvhRef.back()->get(), material});
    }

    void run() {
        auto&& env = Environment::get();
        env.init(AppType::Online);
        {
            Stream resLoader;
            mLight.emplace_back(makeLightWrapper<PointLight>(resLoader, Point{3.0f, 3.0f, 3.0f},
                Spectrum{RGB{10.0f, 20.0f, 30.0f}}));
            mLight.emplace_back(makeLightWrapper<PointLight>(resLoader, Point{-3.0f, 3.0f, 3.0f},
                Spectrum{RGB{30.0f, 20.0f, 10.0f}}));
            mMaterial = MemorySpan<MaterialWrapper>(2);
            const TextureMapping2DWrapper mapping{UVMapping{}};
            {
                const TextureSampler2DSpectrumWrapper samplerS{ConstantSampler2DSpectrum{Spectrum{1.0f}}};
                const TextureSampler2DFloatWrapper samplerF{ConstantSampler2DFloat{0.1f}};
                const Texture2DSpectrum textureS{mapping, samplerS};
                const Texture2DFloat textureF{mapping, samplerF};
                MaterialWrapper plastic{Plastic{textureS, textureS, textureF}};
                checkError(hipMemcpyAsync(mMaterial.begin(), &plastic, sizeof(MaterialWrapper),
                    hipMemcpyHostToDevice, resLoader.get()));
            }
            {
                const TextureSampler2DSpectrumWrapper samplerR{ConstantSampler2DSpectrum{Spectrum{0.0f}}};
                const TextureSampler2DSpectrumWrapper samplerT{ConstantSampler2DSpectrum{Spectrum{1.0f}}};
                const TextureSampler2DFloatWrapper index{ConstantSampler2DFloat{1.5f}};
                const TextureSampler2DFloatWrapper roughness{ConstantSampler2DFloat{0.0f}};
                const Texture2DSpectrum textureR{mapping, samplerR};
                const Texture2DSpectrum textureT{mapping, samplerT};
                const Texture2DFloat indexT{mapping, index};
                const Texture2DFloat roughnessT{mapping, roughness};
                MaterialWrapper glass{Glass{textureR, textureT, indexT, roughnessT, roughnessT}};
                checkError(hipMemcpyAsync(mMaterial.begin() + 1, &glass, sizeof(MaterialWrapper),
                    hipMemcpyHostToDevice, resLoader.get()));
            }

            std::vector<Primitive> primitives;
            const auto cubeMat = glm::scale(glm::rotate(
                glm::translate(glm::mat4{}, {0.0f, 0.0f, 1.0f}), 45.0f, Vector(1.0f)), Vector(1e-3f));
            addModel(resLoader, primitives, cubeMat, "Res/cube.obj", mMaterial.begin() + 1);
            const auto objectMat = glm::scale(glm::mat4{}, Vector(5.0f));
            addModel(resLoader, primitives, objectMat, "Res/dragon.obj", mMaterial.begin());
            std::vector<LightWrapper*> lights;
            for (auto&& light : mLight)
                lights.emplace_back(light.begin());
            mScene = std::make_unique<SceneDesc>(primitives, lights);
            resLoader.sync();
        }
        SequenceGenerator2DWrapper sequenceGenerator{Halton2D{}};
        const SampleWeightLUT lut(64U, FilterWrapper{TriangleFilter{}});
        const uvec2 imageSize{1920U, 1080U};
        mIntegrator = std::make_unique<PathIntegrator>(sequenceGenerator, 10U, 1024U, 256U);
        const auto beg = Clock::now();
        const Transform toCamera{
            glm::lookAt(Vector{0.0f, 0.0f, 2.0f}, Vector{0.0f, 0.0f, 0.0f}, Vector{0.0f, 1.0f, 0.0f})
        };
        mCamera.near = 1.0f;
        auto res = renderFrame(*mIntegrator, *mScene, inverse(toCamera),
            RayGeneratorWrapper(mCamera.getRayGenerator(imageSize)), lut, imageSize, 32U);
        const auto end = Clock::now();
        const auto t = std::chrono::duration_cast<std::chrono::microseconds>(end - beg).count();
        printf("%.3lf ms\n", t * 1e-3);
        PinnedBuffer<Spectrum> pixel(res.size());
        hipMemcpy(pixel.begin(), res.begin(), sizeof(Spectrum) * res.size(), hipMemcpyDeviceToHost);
        std::vector<float> pixelFloat(pixel.size() * 3);
        auto valid = true;
        for (size_t i = 0; i < pixel.size(); ++i) {
            const auto col = pixel[i].toRGB();
            pixelFloat[i * 3] = col.r;
            pixelFloat[i * 3 + 1] = col.g;
            pixelFloat[i * 3 + 2] = col.b;
            valid &= (isfinite(pixel[i].lum()));
        }
        saveHdr("output.hdr", pixelFloat.data(), imageSize);
        if (!valid)printf("The image is invalid.");
        system("pause");
        env.uninit();
    }
};

int main() {
    App app;
    app.run();
    return 0;
}
